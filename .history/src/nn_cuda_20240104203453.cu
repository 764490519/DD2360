#include "hip/hip_runtime.h"
 /*
 * nn.cu
 * Nearest Neighbor
 *
 */

#include <stdio.h>
#include <sys/time.h>
#include <float.h>
#include <vector>
#include "hip/hip_runtime.h"

#define min( a, b )			a > b ? b : a
#define ceilDiv( a, b )		( a + b - 1 ) / b
#define print( x )			printf( #x ": %lu\n", (unsigned long) x )
#define DEBUG				false

#define DEFAULT_THREADS_PER_BLOCK 256

#define MAX_ARGS 10
#define REC_LENGTH 53 // size of a record in db
#define LATITUDE_POS 28	// character position of the latitude value in each record
#define OPEN 10000	// initial value of nearest neighbors
typedef struct latLong
{
  float lat;
  float lng;
} LatLong;

typedef struct record
{
  char recString[REC_LENGTH];
  float distance;
} Record;

int loadData(char *filename,std::vector<Record> &records,std::vector<LatLong> &locations);
void findLowest(std::vector<Record> &records,float *distances,int numRecords,int topN);
void printUsage();
int parseCommandline(int argc, char *argv[], char* filename,int *r,float *lat,float *lng,
                     int *q, int *t, int *p, int *d);
void printLowest(std::vector<Record> &records, int *min_record, int topN, float *dis_min);
/**
* Kernel
* Executed on GPU
* Calculates the Euclidean distance from each record in the database to the target position
*/
__global__ void euclid(LatLong *d_locations, float *d_distances, int numRecords,float lat, float lng)
{
	//int globalId = gridDim.x * blockDim.x * blockIdx.y + blockDim.x * blockIdx.x + threadIdx.x;
	int globalId = blockDim.x * ( gridDim.x * blockIdx.y + blockIdx.x ) + threadIdx.x; // more efficient
    LatLong *latLong = d_locations+globalId;
    if (globalId < numRecords) {
        float *dist=d_distances+globalId;
        *dist = (float)sqrt((lat-latLong->lat)*(lat-latLong->lat)+(lng-latLong->lng)*(lng-latLong->lng));
	}
}

//added by yitong
__device__ void swap(float* a, float* b) {
    float t = *a;
    *a = *b;
    *b = t;
}

/*
partition the array, and return the pivot index
*/
__device__ int partition(float* arr, int low, int high) {
    float pivot = arr[high];
    int i = low;
    for (int j = low; j < high; j++) {
        if (arr[j] < pivot) {
            swap(&arr[i], &arr[j]);
            i++;
        }
    }
    swap(&arr[i], &arr[high]);
    return i;
}
/*
find the min "numMin" elements in the array from "offset_mul" to "offset_mul + offset"
the result is stored in "d_minLoc" from "offset_min" to "offset_min + numMin"
time complexity is O(numRecords/sqrt(numRecords/numMin))
find_min and find_min_final uses the same algorithm, Quickselect
*/

__global__ void find_min( float *d_distances, int numRecords, int *d_minLoc, int offset, int numMin, float *tmp)
{
    int globalId = blockIdx.x * blockDim.x + threadIdx.x;
    int low = 0, high = (globalId + 1)* offset > numRecords ? numRecords - globalId* offset - 1 : offset - 1;
    int max = high, offset_min = numMin * globalId, k = 0, offset_mul = offset * globalId, pivotIndex;
    float min_k;
    if(offset_mul < numRecords){
        float* now = tmp + offset_mul;
        for(int i = 0; i < max; i++){
            now[i] = d_distances[i + offset_mul];
        }
        //quickselect
        while (low <= high) {
            // Partition the array and get the pivot index
            pivotIndex = partition(now, low, high);
           if (pivotIndex == numMin){
                min_k = now[pivotIndex];
                break;
            }   
            // If numMin is less, continue to the left part
            else if (pivotIndex > numMin) high = pivotIndex - 1;

            // If numMin is more, continue to the right part
            else low = pivotIndex + 1;
        }
        //get the min "numMin" elements
        for(int i = 0; i < max; i++){
            if(d_distances[i+offset_mul] < min_k){
                d_minLoc[offset_min + k] = i + offset_mul;
                k++;
            }
        }
        //if there are elements equal in min_k, get them 
        if(k < numMin){
            for(int i = 0; i < max; i++){
                if((d_distances[i+offset_mul] == min_k) && k < numMin){
                    d_minLoc[offset_min + k] = i + offset_mul;
                    k++;
                }
            }
        }      
    }
  //printf("\n");
}
/*
find the min "numMin" element on the base of the result of "find_min"
the result is stored in "d_minLoc" from "0" to "numMin"
time complexity is O(sqrt(numRecords/numMin) * numMin)
*/
__global__ void find_min_final(float *d_distances, int num, int *d_minLoc, int numMin, float *min_dis, float *dis_min, int *d_minmem)
{
    float min_k = 0;
    int pivotIndex;
    int low = 0, high = num - 1, k = 0;
    for(int i = 0; i < num; i++)
    {
        dis_min[i] = d_distances[d_minLoc[i]];
    }
    //quickselect
    while (low <= high) {
        // Partition the array and get the pivot index
        pivotIndex = partition(dis_min, low, high);
         // If pivot itself is the kth smallest element
        if (pivotIndex == numMin){
            min_k = dis_min[pivotIndex];
            break;
        }   
         // If k is less, continue to the left part
        else if (pivotIndex > numMin) high = pivotIndex - 1;

         // If k is more, continue to the right part
        else low = pivotIndex + 1;
    }
    //get the min "numMin" elements
    for(int i = 0; i < num; i++)
    {
        if(d_distances[d_minLoc[i]] < min_k)
        {
          d_minmem[k] = d_minLoc[i];
          printf("d_minmem[%d] :%d\n", i,d_minmem[k]);
          k++;
        }
    }
    //if there are elements equal in min_k, get them
    if(k < numMin){
        for(int i = 0; i < num; i++){
          if((d_distances[d_minLoc[i]] == min_k )&& (k < numMin)){
            d_minmem[k] = d_minLoc[i];
          printf("d_minmem[%d] :%d\n", i,d_minmem[k]);
            k++;
          }
        }
      }     
    for(int i = 0; i< numMin; i++){
      min_dis[i] = d_distances[d_minmem[i]];
      printf("min_dis[%d] :%f\n", i,min_dis[i]);
    }
}
double cpuSecond() {
   hipDeviceSynchronize();
   struct timeval tp;
   gettimeofday(&tp,NULL);
   return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}

/**
* This program finds the k-nearest neighbors
**/

int main(int argc, char* argv[])
{
	float lat, lng;
	int quiet=0,timing=0,platform=0,device=0;

  std::vector<Record> records;
	std::vector<LatLong> locations;
	char filename[100];
	int resultsCount=10;
  float *dis_min , *tmp;

    // parse command line
    if (parseCommandline(argc, argv, filename,&resultsCount,&lat,&lng,
                     &quiet, &timing, &platform, &device)) {
      printUsage();
      return 0;
    }

    int numRecords = loadData(filename,records,locations);
    if (resultsCount > numRecords) resultsCount = numRecords;

    //for(i=0;i<numRecords;i++)
    //  printf("%s, %f, %f\n",(records[i].recString),locations[i].lat,locations[i].lng);


    //Pointers to host memory
	//float *distances;
	//Pointers to device memory
	LatLong *d_locations;
	float *d_distances, *min_record_dis, *tmp_final_float;
  int *d_minLoc, *min_record, *d_minmem;


	// Scaling calculations - added by Sam Kauffman
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties( &deviceProp, 0 );
	hipDeviceSynchronize();
	unsigned long maxGridX = deviceProp.maxGridSize[0];
	unsigned long threadsPerBlock = min( deviceProp.maxThreadsPerBlock, DEFAULT_THREADS_PER_BLOCK );
	size_t totalDeviceMemory;
	size_t freeDeviceMemory;
	hipMemGetInfo(  &freeDeviceMemory, &totalDeviceMemory );
	hipDeviceSynchronize();
	unsigned long usableDeviceMemory = freeDeviceMemory * 85 / 100; // 85% arbitrary throttle to compensate for known CUDA bug
	unsigned long maxThreads = usableDeviceMemory / 12; // 4 bytes in 3 vectors per thread
	if ( numRecords > maxThreads )
	{
		fprintf( stderr, "Error: Input too large.\n" );
		exit( 1 );
	}
	unsigned long blocks = ceilDiv( numRecords, threadsPerBlock ); // extra threads will do nothing
	unsigned long gridY = ceilDiv( blocks, maxGridX );
	unsigned long gridX = ceilDiv( blocks, gridY );
	// There will be no more than (gridY - 1) extra blocks
	dim3 gridDim( gridX, gridY );
  dim3 grid_min(((int)(sqrt((float)numRecords/(float)resultsCount)+1) + threadsPerBlock - 1)/threadsPerBlock);
	if ( DEBUG )
	{
		print( totalDeviceMemory ); // 804454400
		print( freeDeviceMemory );
		print( usableDeviceMemory );
		print( maxGridX ); // 65535
		print( deviceProp.maxThreadsPerBlock ); // 1024
		print( threadsPerBlock );
		print( maxThreads );
		print( blocks ); // 130933
		print( gridY );
		print( gridX );
	}

	/**
	* Allocate memory on host and device
	*/
	//distances = (float *)malloc(sizeof(float) * numRecords);
  min_record = (int *)malloc(sizeof(int) * resultsCount);
  dis_min = (float *)malloc(sizeof(float) * resultsCount);
	hipMalloc((void **) &d_locations,sizeof(LatLong) * numRecords);
	hipMalloc((void **) &d_distances,sizeof(float) * numRecords);
  hipMalloc((void **) &d_minLoc,sizeof(int) * (int)(sqrt((float)numRecords/(float)resultsCount)+1)* resultsCount);
  hipMalloc((void **) &min_record_dis,sizeof(float) * resultsCount);
  hipMalloc((void **) &tmp,sizeof(float) * numRecords);
  hipMalloc((void **) &tmp_final_float,sizeof(float) * (int)(sqrt((float)numRecords/(float)resultsCount)+1)* resultsCount);
  hipMalloc((void **) &d_minmem,sizeof(int) * resultsCount);
   /**
    * Transfer data from host to device
    */
    hipMemcpy( d_locations, &locations[0], sizeof(LatLong) * numRecords, hipMemcpyHostToDevice);
    /**
    * Execute kernel
    */
    euclid<<< gridDim, threadsPerBlock >>>(d_locations,d_distances,numRecords,lat,lng);
    hipDeviceSynchronize();
    int offset =(float)numRecords / (int)(sqrt((float)numRecords/(float)resultsCount)+1) + 1;
    find_min<<< grid_min , threadsPerBlock >>>(d_distances, numRecords, d_minLoc, offset, resultsCount,tmp);
    hipDeviceSynchronize();
    find_min_final<<< 1, 1 >>>(d_distances, (int)(sqrt((float)numRecords/(float)resultsCount)+1) * resultsCount, d_minLoc, resultsCount, min_record_dis, tmp_final_float, d_minmem);
    hipDeviceSynchronize();
    //Copy data from device memory to host memory
    //hipMemcpy( distances, d_distances, sizeof(float)*numRecords, hipMemcpyDeviceToHost );
    hipMemcpy( min_record, d_minmem, sizeof(int)*resultsCount, hipMemcpyDeviceToHost );
    hipMemcpy( dis_min, min_record_dis, sizeof(float)*resultsCount, hipMemcpyDeviceToHost );
	// find the resultsCount least distances
    //findLowest(records,distances,numRecords,resultsCount);
    printLowest(records, min_record, resultsCount, dis_min);
    // print out results
    /*if (!quiet)
    for(i=0;i<resultsCount;i++) {
      printf("%s --> Distance=%f\n",records[i].recString,records[i].distance);
    }*/
    free(min_record);
    free(dis_min);
    //Free memory
	hipFree(d_locations);
	hipFree(d_distances);
  hipFree(d_minLoc);
  hipFree(min_record_dis);
  hipFree(tmp);
  hipFree(tmp_final_float);
  hipFree(d_minmem);
    return 0;

}

int loadData(char *filename,std::vector<Record> &records,std::vector<LatLong> &locations){
    FILE   *flist,*fp;
	int    i=0;
	char dbname[64];
	int recNum=0;

    /**Main processing **/

    flist = fopen(filename, "r");
	while(!feof(flist)) {
		/**
		* Read in all records of length REC_LENGTH
		* If this is the last file in the filelist, then done
		* else open next file to be read next iteration
		*/
		if(fscanf(flist, "%s\n", dbname) != 1) {
            fprintf(stderr, "error reading filelist\n");
            exit(0);
        }
        fp = fopen(dbname, "r");
        if(!fp) {
            printf("error opening a db\n");
            exit(1);
        }
        // read each record
        while(!feof(fp)){
            Record record;
            LatLong latLong;
            fgets(record.recString,49,fp);
            fgetc(fp); // newline
            if (feof(fp)) break;

            // parse for lat and long
            char substr[6];

            for(i=0;i<5;i++) substr[i] = *(record.recString+i+28);
            substr[5] = '\0';
            latLong.lat = atof(substr);

            for(i=0;i<5;i++) substr[i] = *(record.recString+i+33);
            substr[5] = '\0';
            latLong.lng = atof(substr);

            locations.push_back(latLong);
            records.push_back(record);
            recNum++;
        }
        fclose(fp);
    }
    fclose(flist);
//    for(i=0;i<rec_count*REC_LENGTH;i++) printf("%c",sandbox[i]);
    return recNum;
}

void findLowest(std::vector<Record> &records,float *distances,int numRecords,int topN){
  int i,j;
  float val;
  int minLoc;
  Record *tempRec;
  float tempDist;

  for(i=0;i<topN;i++) {
    minLoc = i;
    for(j=i;j<numRecords;j++) {
      val = distances[j];
      if (val < distances[minLoc]) minLoc = j;
    }
    // swap locations and distances
    tempRec = &records[i];
    records[i] = records[minLoc];
    records[minLoc] = *tempRec;

    tempDist = distances[i];
    distances[i] = distances[minLoc];
    distances[minLoc] = tempDist;

    // add distance to the min we just found
    records[i].distance = distances[i];
  }
}
void printLowest(std::vector<Record> &records, int *min_record, int topN, float *dis_min){
  for(int i = 0; i < topN; i++)
    printf("%s --> Distance=%f\n",records[min_record[i]].recString,dis_min[i]);
}
int parseCommandline(int argc, char *argv[], char* filename,int *r,float *lat,float *lng,
                     int *q, int *t, int *p, int *d){
    int i;
    if (argc < 2) return 1; // error
    strncpy(filename,argv[1],100);
    char flag;

    for(i=1;i<argc;i++) {
      if (argv[i][0]=='-') {// flag
        flag = argv[i][1];
          switch (flag) { 
            case 'r': // number of results
              i++;
              *r = atoi(argv[i]);
              break;
            case 'l': // lat or lng
              if (argv[i][2]=='a') {//lat
                *lat = atof(argv[i+1]);
              }
              else {//lng
                *lng = atof(argv[i+1]);
              }
              i++;
              break;
            case 'h': // help
              return 1;
            case 'q': // quiet
              *q = 1;
              break;
            case 't': // timing
              *t = 1;
              break;
            case 'p': // platform
              i++;
              *p = atoi(argv[i]);
              break;
            case 'd': // device
              i++;
              *d = atoi(argv[i]);
              break;
        }
      }
    }
    if ((*d >= 0 && *p<0) || (*p>=0 && *d<0)) // both p and d must be specified if either are specified
      return 1;
    return 0;
}

void printUsage(){
  printf("Nearest Neighbor Usage\n");
  printf("\n");
  printf("nearestNeighbor [filename] -r [int] -lat [float] -lng [float] [-hqt] [-p [int] -d [int]]\n");
  printf("\n");
  printf("example:\n");
  printf("$ ./nearestNeighbor filelist.txt -r 5 -lat 30 -lng 90\n");
  printf("\n");
  printf("filename     the filename that lists the data input files\n");
  printf("-r [int]     the number of records to return (default: 10)\n");
  printf("-lat [float] the latitude for nearest neighbors (default: 0)\n");
  printf("-lng [float] the longitude for nearest neighbors (default: 0)\n");
  printf("\n");
  printf("-h, --help   Display the help file\n");
  printf("-q           Quiet mode. Suppress all text output.\n");
  printf("-t           Print timing information.\n");
  printf("\n");
  printf("-p [int]     Choose the platform (must choose both platform and device)\n");
  printf("-d [int]     Choose the device (must choose both platform and device)\n");
  printf("\n");
  printf("\n");
  printf("Notes: 1. The filename is required as the first parameter.\n");
  printf("       2. If you declare either the device or the platform,\n");
  printf("          you must declare both.\n\n");
}
